#include "hip/hip_runtime.h"
/*
 * This file is part of Co-Section.
 *
 * Copyright (C) 2020 Max-Planck-Gesellschaft.
 * Developed by Michael Strecke <mstrecke at tue dot mpg dot de>.
 * For more information see <https://cosection.is.tue.mpg.de/>.
 * If you use this code, please cite the respective publication as
 * listed on the website.
 */
#include "CoSection/optim/cuda/OptSDF.cuh"

#include <iomanip>

// using namespace emf::cuda;
using emf::cuda::float33;
using emf::cuda::operator+;
using emf::cuda::operator-;
using emf::cuda::operator/;
using emf::cuda::dot;
using emf::cuda::norm;

namespace cosection {
namespace cuda {
namespace OptSDF {

__global__
void kernel_compSDFWeights ( cv::cuda::PtrStepSz<float3> points,
                             cv::cuda::PtrStep<float3> normals,
                             cv::cuda::PtrStep<float> assocW,
                             cv::cuda::PtrStep<float> ws,
                             cv::cuda::PtrStep<float> cs,
                             cv::cuda::PtrStep<float> wcounts,
                             cv::cuda::PtrStep<float> ds, float sigma,
                             float33 rot, float3 trans, int3 volSize,
                             float voxelSize ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= points.cols )
        return;

    const float3 p = points ( 0, i );
    const float3 n = normals ( 0, i );
    const float assoc = assocW ( 0, i );

    const float3 p_o = rot * p + trans;
    const float3 n_o = rot * n;

    const int3 volIdx = make_int3 (
                            ( p_o.x / voxelSize ) + ( volSize.x - 1 ) / 2.f,
                            ( p_o.y / voxelSize ) + ( volSize.y - 1 ) / 2.f,
                            ( p_o.z / voxelSize ) + ( volSize.z - 1 ) / 2.f
                        );

    for ( int x = max ( ( int ) ( volIdx.x - 3 * sigma/voxelSize ), 0 );
            x < min ( ( int ) ( volIdx.x + 3 * sigma/voxelSize ), volSize.x );
            ++x ) {
        for ( int y = max ( ( int ) ( volIdx.y - 3 * sigma/voxelSize ), 0 );
                y < min ( ( int ) ( volIdx.y + 3 * sigma/voxelSize ),
                          volSize.y );
                ++y ) {
            for ( int z = max ( ( int ) ( volIdx.z - 3 * sigma/voxelSize ), 0 );
                    z < min ( ( int ) ( volIdx.z + 3 * sigma/voxelSize ),
                              volSize.z );
                    ++z ) {
                if ( ds ( z * volSize.y + y, x ) <= 0.f ) {
                    float3 v = make_float3 (
                                   ( x - ( volSize.x - 1 ) / 2.f ) * voxelSize,
                                   ( y - ( volSize.y - 1 ) / 2.f ) * voxelSize,
                                   ( z - ( volSize.z - 1 ) / 2.f ) * voxelSize
                               );
                    float w = exp ( - ( ( p_o.x - v.x ) * ( p_o.x - v.x )
                                        + ( p_o.y - v.y ) * ( p_o.y - v.y )
                                        + ( p_o.z - v.z ) * ( p_o.z - v.z ) )
                                    / ( sigma*sigma ) ) * assoc;
                    float c = w * dot ( v - p_o, n_o );

                    atomicAdd ( &ws ( z * volSize.y + y, x ), w );
                    atomicAdd ( &cs ( z * volSize.y + y, x ), c );
                    atomicAdd ( &wcounts ( z * volSize.y + y, x ), 1.f );
                }
            }
        }
    }
}

void compSDFWeights ( const cv::cuda::GpuMat& points,
                      const cv::cuda::GpuMat& normals,
                      const cv::cuda::GpuMat& assocW, cv::cuda::GpuMat& w,
                      cv::cuda::GpuMat& c, cv::cuda::GpuMat& wcount,
                      cv::cuda::GpuMat& d, float sigma,
                      const cv::Matx33f& rot_WO, const cv::Vec3f& trans_WO,
                      const cv::Vec3i& volumeRes, const float voxelSize ) {
    dim3 threads ( 1024 );
    dim3 blocks ( ( points.cols + threads.x - 1 ) / threads.x );

    const float33 rot = * ( float33 * ) rot_WO.val;
    const float3 trans = * ( float3 * ) trans_WO.val;
    const int3 volSize = * ( int3 * ) volumeRes.val;

    kernel_compSDFWeights<<<blocks, threads>>> (
        points, normals, assocW, w, c, wcount, d, sigma, rot, trans, volSize,
        voxelSize );

    hipDeviceSynchronize();
}

__global__
void kernel_delObjFg ( const cv::cuda::PtrStepSz<bool> probs,
                       cv::cuda::PtrStep<float> w, cv::cuda::PtrStep<float> c,
                       cv::cuda::PtrStep<float> wcount, const float33 rot,
                       const float3 trans, const float thisVoxelSize,
                       const float otherVoxelSize, const int3 thisRes,
                       const int3 otherRes ) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y_ = blockIdx.y * blockDim.y + threadIdx.y;

    if ( x >= probs.cols || y_ >= probs.rows || !probs ( y_, x ) )
        return;

    const int y = y_ % thisRes.y;
    const int z = y_ / thisRes.y;

    const float3 v = make_float3 (
                         ( x - ( thisRes.x - 1 ) / 2.f ) * thisVoxelSize,
                         ( y - ( thisRes.y - 1 ) / 2.f ) * thisVoxelSize,
                         ( z - ( thisRes.z - 1 ) / 2.f ) * thisVoxelSize );
    const float3 v_other = rot * v + trans;
    const float3 idx_other = ( v_other / otherVoxelSize )
                             + ( otherRes - 1 ) / 2.f;
    if ( idx_other.x < 0 || idx_other.x >= otherRes.x - 1
            || idx_other.y < 0 || idx_other.y >= otherRes.y - 1
            || idx_other.z < 0 || idx_other.z >= otherRes.z - 1 )
        return;

    const int3 lowIdx = make_int3 ( static_cast<int> ( idx_other.x ),
                                    static_cast<int> ( idx_other.y ),
                                    static_cast<int> ( idx_other.z ) );

    w ( lowIdx.z * otherRes.y + lowIdx.y, lowIdx.x )                 = 0.f;
    w ( lowIdx.z * otherRes.y + lowIdx.y, lowIdx.x + 1 )             = 0.f;
    w ( lowIdx.z * otherRes.y + lowIdx.y + 1, lowIdx.x )             = 0.f;
    w ( lowIdx.z * otherRes.y + lowIdx.y + 1, lowIdx.x + 1 )         = 0.f;
    w ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y, lowIdx.x )         = 0.f;
    w ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y, lowIdx.x + 1 )     = 0.f;
    w ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y + 1, lowIdx.x )     = 0.f;
    w ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y + 1, lowIdx.x + 1 ) = 0.f;

    c ( lowIdx.z * otherRes.y + lowIdx.y, lowIdx.x )                 = 0.f;
    c ( lowIdx.z * otherRes.y + lowIdx.y, lowIdx.x + 1 )             = 0.f;
    c ( lowIdx.z * otherRes.y + lowIdx.y + 1, lowIdx.x )             = 0.f;
    c ( lowIdx.z * otherRes.y + lowIdx.y + 1, lowIdx.x + 1 )         = 0.f;
    c ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y, lowIdx.x )         = 0.f;
    c ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y, lowIdx.x + 1 )     = 0.f;
    c ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y + 1, lowIdx.x )     = 0.f;
    c ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y + 1, lowIdx.x + 1 ) = 0.f;

    wcount ( lowIdx.z * otherRes.y + lowIdx.y, lowIdx.x )                 = 0.f;
    wcount ( lowIdx.z * otherRes.y + lowIdx.y, lowIdx.x + 1 )             = 0.f;
    wcount ( lowIdx.z * otherRes.y + lowIdx.y + 1, lowIdx.x )             = 0.f;
    wcount ( lowIdx.z * otherRes.y + lowIdx.y + 1, lowIdx.x + 1 )         = 0.f;
    wcount ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y, lowIdx.x )         = 0.f;
    wcount ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y, lowIdx.x + 1 )     = 0.f;
    wcount ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y + 1, lowIdx.x )     = 0.f;
    wcount ( ( lowIdx.z + 1 ) * otherRes.y + lowIdx.y + 1, lowIdx.x + 1 ) = 0.f;
}

void delObjFg ( const cv::cuda::GpuMat& probs, cv::cuda::GpuMat& w,
                cv::cuda::GpuMat& c, cv::cuda::GpuMat& wcount,
                const cv::Matx33f& rel_rot, const cv::Vec3f& rel_trans,
                const float thisVoxelSize, const float otherVoxelSize,
                const cv::Vec3i& thisVolumeRes,
                const cv::Vec3i& otherVolumeRes ) {
    dim3 threads ( 16, 16 );
    dim3 blocks ( ( probs.cols + threads.x - 1 ) / threads.x,
                  ( probs.rows + threads.y - 1 ) / threads.y );

    const float33 rot = * ( float33 * ) rel_rot.val;
    const float3 trans = * ( float3 * ) rel_trans.val;

    const int3 thisRes = * ( int3 * ) thisVolumeRes.val;
    const int3 otherRes = * ( int3 * ) otherVolumeRes.val;

    kernel_delObjFg<<<blocks, threads>>> ( probs, w, c, wcount, rot,
                                           trans, thisVoxelSize, otherVoxelSize,
                                           thisRes, otherRes );
    hipDeviceSynchronize();
}

inline __device__
float Dxx ( const float* u, const int idx, const int step, const int gidx,
            const int gstep, const int limit ) {
    if ( ( gidx / gstep ) % limit > 0 && ( gidx / gstep ) % limit < limit - 1 )
        return u[idx - step] - 2 * u[idx] + u[idx + step];
    else
        return 0.f;
}

inline __device__
float DxxT ( const float* u, const int idx, const int step, const int gidx,
             const int gstep, const int limit ) {
    return Dxx ( u, idx - step, step, gidx - gstep, gstep, limit )
           - 2 * Dxx ( u, idx, step, gidx, gstep, limit )
           + Dxx ( u, idx + step, step, gidx + gstep, gstep, limit );
}

inline __device__
float Dxy ( const float* u, const int idx, const int xstep,
            const int ystep, const int x, const int y, const int maxX,
            const int maxY ) {
    const int xshiftm = x > 0 ? 1 : 0;
    const int xshiftp = x < maxX - 1 ? 1 : 0;
    const int yshiftm = y > 0 ? 1 : 0;
    const int yshiftp = y < maxY - 1 ? 1 : 0;

    float ret = u[idx - xstep * xshiftm - ystep * yshiftm]
                - u[idx + xstep * xshiftp - ystep * yshiftm]
                - u[idx - xstep * xshiftm + ystep * yshiftp]
                + u[idx + xstep * xshiftp + ystep * yshiftp];
    if ( x > 0 && x < maxX - 1 )
        ret /= 2.f;
    if ( y > 0 && y < maxY - 1 )
        ret /= 2.f;

    return ret;
}

inline __device__
float DxyT ( const float* u, const int idx, const int xstep,
             const int ystep, const int x, const int y, const int maxX,
             const int maxY ) {
    float dxyt = 0.f;
    char signconfig = 0b1001; // 1: plus, 0: minus
    if ( x == 0 )
        signconfig ^= 0b0101;
    else if ( x == maxX - 1 )
        signconfig ^= 0b1010;

    if ( y == 0 )
        signconfig ^= 0b0011;
    else if ( y == maxY - 1 )
        signconfig ^= 0b1100;

    for ( int i = 0; i < 4; ++i ) {
        int xshift = -1 + ( ( i & 1 ) << 1 );
        xshift = x + xshift >= 0 && x + xshift < maxX ? xshift : 0;
        int yshift = -1 + ( i & 2 );
        yshift = y + yshift >= 0 && y + yshift < maxY ? yshift : 0;

        float temp = Dxy ( u, idx + xstep * xshift + ystep * yshift,
                           xstep, ystep, x + xshift, y + yshift, maxX, maxY );
        if ( x + xshift > 0 && x + xshift < maxX - 1 )
            temp /= 2.f;
        if ( y + yshift > 0 && y + yshift < maxY - 1 )
            temp /= 2.f;

        if ( ( signconfig >> i ) & 1 )
            dxyt += temp;
        else
            dxyt -= temp;
    }
    return dxyt;
}

__global__
void kernel_multB ( const cv::cuda::PtrStepSz<float> u,
                    const cv::cuda::PtrStep<float> w,
                    const cv::cuda::PtrStep<float> dHull,
                    const cv::cuda::PtrStep<float> dInter,
                    cv::cuda::PtrStep<float> u_new, const int3 volSize,
                    const float voxelSize, const float alpha,
                    const float betaHull, const float betaInter ) {
    extern __shared__ float data[];

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    if ( x < 0 || x >= volSize.x || y < 0 || y >= volSize.y
            || z < 0 || z >= volSize.z )
        return;

    const int bidx = ( ( threadIdx.z + 2 ) * ( blockDim.y + 4 )
                       + threadIdx.y + 2 ) * ( blockDim.x + 4 )
                     + threadIdx.x + 2;
    const int idx = ( z * volSize.y + y ) * volSize.x + x;

    // Load data from global to shared mem
    data[bidx] = u.ptr() [idx];
    if ( threadIdx.x < 2 && x > 2 ) {
        data[bidx - 2] = u.ptr() [idx - 2];

        if ( threadIdx.y < 2 && y > 2 )
            data[bidx - 2 - 2 * ( blockDim.x + 4 )] =
                u.ptr() [idx - 2 - 2 * volSize.x];
        else if ( threadIdx.y >= blockDim.y - 2 && y + 2 < volSize.y )
            data[bidx - 2 + 2 * ( blockDim.x + 4 )] =
                u.ptr() [idx - 2 + 2 * volSize.x];
        if ( threadIdx.z < 2 && z > 2 )
            data[bidx - 2 - 2 * ( blockDim.x + 4 ) * ( blockDim.y + 4 )] =
                u.ptr() [idx - 2 - 2 * volSize.x * volSize.y];
        else if ( threadIdx.z >= blockDim.z - 2 && z + 2 < volSize.z )
            data[bidx - 2 + 2 * ( blockDim.x + 4 ) * ( blockDim.y + 4 )] =
                u.ptr() [idx - 2 + 2 * volSize.x * volSize.y];
    } else if ( threadIdx.x >= blockDim.x - 2 && x + 2 < volSize.x ) {
        data[bidx + 2] = u.ptr() [idx + 2];

        if ( threadIdx.y < 2 && y > 2 )
            data[bidx + 2 - 2 * ( blockDim.x + 4 )] =
                u.ptr() [idx + 2 - 2 * volSize.x];
        else if ( threadIdx.y >= blockDim.y - 2 && y + 2 < volSize.y )
            data[bidx + 2 + 2 * ( blockDim.x + 4 )] =
                u.ptr() [idx + 2 + 2 * volSize.x];
        if ( threadIdx.z < 2 && z > 2 )
            data[bidx + 2 - 2 * ( blockDim.x + 4 ) * ( blockDim.y + 4 )] =
                u.ptr() [idx + 2 - 2 * volSize.x * volSize.y];
        else if ( threadIdx.z >= blockDim.z - 2 && z + 2 < volSize.z )
            data[bidx + 2 + 2 * ( blockDim.x + 4 ) * ( blockDim.y + 4 )] =
                u.ptr() [idx + 2 + 2 * volSize.x * volSize.y];
    }

    if ( threadIdx.y < 2 && y > 2 ) {
        data[bidx - 2 * ( blockDim.x + 4 )] = u.ptr() [idx - 2 * volSize.x];

        if ( threadIdx.z < 2 && z > 2 )
            data[bidx - 2 * ( blockDim.x + 4 )
                      - 2 * ( blockDim.x + 4 ) * ( blockDim.y + 4 )] =
                     u.ptr() [idx - 2 * volSize.x - 2 * volSize.x * volSize.y];
        else if ( threadIdx.z >= blockDim.z - 2 && z + 2 < volSize.z )
            data[bidx - 2 * ( blockDim.x + 4 )
                      + 2 * ( blockDim.x + 4 ) * ( blockDim.y + 4 )] =
                     u.ptr() [idx - 2 * volSize.x + 2 * volSize.x * volSize.y];
    } else if ( threadIdx.y >= blockDim.y - 2 && y + 2 < volSize.y ) {
        data[bidx + 2 * ( blockDim.x + 4 )] = u.ptr() [idx + 2 * volSize.x];

        if ( threadIdx.z < 2 && z > 2 )
            data[bidx + 2 * ( blockDim.x + 4 )
                      - 2 * ( blockDim.x + 4 ) * ( blockDim.y + 4 )] =
                     u.ptr() [idx + 2 * volSize.x - 2 * volSize.x * volSize.y];
        else if ( threadIdx.z >= blockDim.z - 2 && z + 2 < volSize.z )
            data[bidx + 2 * ( blockDim.x + 4 )
                      + 2 * ( blockDim.x + 4 ) * ( blockDim.y + 4 )] =
                     u.ptr() [idx + 2 * volSize.x + 2 * volSize.x * volSize.y];
    }

    if ( threadIdx.z < 2 && z > 2 )
        data[bidx - 2 * ( blockDim.x + 4 ) * ( blockDim.y + 4 )] =
            u.ptr() [idx - 2 * volSize.x * volSize.y];
    else if ( threadIdx.z >= blockDim.z - 2 && z + 2 < volSize.z )
        data[bidx + 2 * ( blockDim.x + 4 ) * ( blockDim.y + 4 )] =
            u.ptr() [idx + 2 * volSize.x * volSize.y];
    __syncthreads();

    const float dxxu = DxxT ( data, bidx, 1, idx, 1, volSize.x );
    const float dyyu = DxxT ( data, bidx, ( blockDim.x + 4 ), idx,
                              volSize.x, volSize.y );
    const float dzzu = DxxT ( data, bidx,
                              ( blockDim.x + 4 ) * ( blockDim.y + 4 ), idx,
                              volSize.x * volSize.y, volSize.z );

    const float dxyu = DxyT ( data, bidx, 1, ( blockDim.x + 4 ), x, y,
                              volSize.x, volSize.y );
    const float dxzu = DxyT ( data, bidx, 1,
                              ( blockDim.x + 4 ) * ( blockDim.y + 4 ), x, z,
                              volSize.x, volSize.z );
    const float dyzu = DxyT ( data, bidx, ( blockDim.x + 4 ),
                              ( blockDim.x + 4 ) * ( blockDim.y + 4 ), y, z,
                              volSize.y, volSize.z );

    const int y_ = idx / u.cols;

    const float hulldist = dHull ( y_, x );
    const float interdist = dInter ( y_, x );
    const float prev_u = u ( y_, x );

    float res = w ( y_, x ) * prev_u
                + alpha // /(voxelSize*voxelSize*voxelSize*voxelSize)
                * ( dxxu + dyyu + dzzu + 2 * ( dxyu + dxzu + dyzu ) );
    if ( hulldist > 0 && hulldist - prev_u > 0 )
        res += betaHull * ( prev_u - hulldist );
    if ( interdist > 0 && interdist - prev_u > 0 )
        res += betaInter * ( prev_u - interdist );

    u_new ( y_, x ) = res;
}

inline __device__
float DxxTDiag ( const int idx, const int limit ) {
    if ( idx > 1 && idx < limit - 2 )
        return 6.f;
    else if ( idx == 1 || idx == limit - 2 )
        return 5.f;
    else
        return 1.f;
}


inline __device__
float DxyTDiag ( const int x, const int y, const int maxX, const int maxY ) {
    if ( x > 1 && x < maxX - 2 && y > 1 && y < maxY - 2 )
        return .25f; // 4 * (1/4)^2 = 1/4 = .25f
    else if ( ( ( x <= 1 || x >= maxX - 2 ) && ( y > 1 && y < maxY - 2 ) )
              || ( ( y <= 1 || y >= maxY - 2 ) && ( x > 1 && x < maxX - 2 ) ) )
        return .625f; // 2 * (1/2)^2 + 2 * (1/4)^2 = 1/2 + 1/8 = 5/8 = .625f
    else
        return 1.5625f; // 1 + 2*(1/2)^2 + (1/4)^2 = 1 + 1/2 + 1/16 = 25/16 = 1.5625f
}

__global__
void kernel_multDinv ( const cv::cuda::PtrStep<float> u_old,
                       cv::cuda::PtrStepSz<float> u,
                       const cv::cuda::PtrStep<float> w,
                       const cv::cuda::PtrStep<float> dHull,
                       const cv::cuda::PtrStep<float> dInter,
                       const int3 volSize, const float voxelSize,
                       const float alpha, const float betaHull,
                       const float betaInter ) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y_ = blockIdx.y * blockDim.y + threadIdx.y;

    if ( x >= u.cols || y_ >= u.rows )
        return;

    const int y = y_ % volSize.y;
    const int z = y_ / volSize.y;

    const float dxxt = DxxTDiag ( x, volSize.x );
    const float dyyt = DxxTDiag ( y, volSize.y );
    const float dzzt = DxxTDiag ( z, volSize.z );

    const float dxyt = DxyTDiag ( x, y, volSize.x, volSize.y );
    const float dxzt = DxyTDiag ( x, z, volSize.x, volSize.z );
    const float dyzt = DxyTDiag ( y, z, volSize.y, volSize.z );

    const float hulldist = dHull ( y_, x );
    const float interdist = dInter ( y_, x );
    const float prev_u = u_old ( y_, x );

    float div = w ( y_, x )
                + alpha // /(voxelSize*voxelSize*voxelSize*voxelSize)
                * ( dxxt + dyyt + dzzt + 2 * ( dxyt + dxzt + dyzt ) );
//     if ( hulldist > 0 && hulldist - prev_u > 0 )
//         div += betaHull * ( 1.f - hulldist / ( prev_u + 1e-6 ) );
//     if ( interdist > 0 && interdist - prev_u > 0 )
//         div += betaInter * ( 1.f - interdist / ( prev_u + 1e-6 ) );

    u ( y_, x ) /= div;
}

void optimizeSDF ( cv::cuda::GpuMat& sdf, const cv::cuda::GpuMat& w,
                   const cv::cuda::GpuMat& c, const cv::cuda::GpuMat& dHull,
                   const cv::cuda::GpuMat& dInter, cv::cuda::GpuMat& buf1,
                   cv::cuda::GpuMat& buf2, const cv::Vec3i& volumeRes,
                   const float voxelSize, const float alpha,
                   const float betaHull, const float betaInter,
                   const int cycleLength ) {
    dim3 threads3 ( 8, 8, 8 );
    dim3 blocks3 ( ( volumeRes[0] + threads3.x - 1 ) / threads3.x,
                   ( volumeRes[1] + threads3.y - 1 ) / threads3.y,
                   ( volumeRes[2] + threads3.z - 1 ) / threads3.z );
    dim3 threads ( 16, 32 );
    dim3 blocks ( ( sdf.cols + threads.x - 1 ) / threads.x,
                  ( sdf.rows + threads.y - 1 ) / threads.y );

    int3 volSize = * ( int3 * ) volumeRes.val;

    createContinuous ( sdf.rows, sdf.cols, CV_32FC1, buf1 );
    createContinuous ( sdf.rows, sdf.cols, CV_32FC1, buf2 );

    // This commented code computes the largest eigenvalue mu of the linear
    // problem. With this computation, omega below could be set to 2/mu.
    // However, we found this not to work well in the experiments and
    // empirically set the value as below.
//     auto begItr = GpuMatBeginItr<float>( buf1 );
//     auto endItr = GpuMatEndItr<float>( buf1 );
//     bool converged;
//     float mu;
//     do {
//         thrust::transform ( thrust::make_counting_iterator(0),
//                             thrust::make_counting_iterator(buf1.cols * buf1.rows),
//                             begItr, prg ( -1, 1 ) );
//         hipDeviceSynchronize();
//
//         double mu_old = cuda::norm ( buf1, NORM_L2 );
//         cuda::divide( buf1, mu_old, buf1 );
//
//         converged = false;
//         int iter;
//         for ( iter = 0, mu = 0; iter < 1000 && !converged; ++iter ) {
//             kernel_multB<<<blocks3, threads3, ( threads3.x + 4 ) * ( threads3.y + 4 ) * ( threads3.z + 4 ) * sizeof(float)>>> ( buf1, w, dHull, dInter, buf2, volSize, alpha, betaHull, betaInter );
//             hipDeviceSynchronize();
//             kernel_multDinv<<<blocks, threads>>> ( buf1, buf2, w, dHull, dInter, volSize, alpha, betaHull, betaInter );
//             hipDeviceSynchronize();
//             mu = cuda::norm( buf2, NORM_L2 );
//     //         std::cout << i << ": " << mu_old << ", " << mu << std::endl;
//             if ( abs ( mu - mu_old ) < 1e-6 ) {
//                 converged = true;
//                 break;
//             }
//             cuda::divide( buf2, mu, buf1 );
//             mu_old = mu;
//         }
//         if ( converged )
//             std::cout << "Eigenvalue computation converged after " << iter << " iterations with mu = " << mu << std::endl;
//         else
//             std::cout << "Eigenvalue computation did not converge!" << std::endl;
//     } while ( !converged );


    float omega = 0.3f;
    std::vector<float> omegas ( cycleLength );
    for ( int i = 0; i < cycleLength; ++i ) {
        float cosval = cos ( M_PI * ( 2 * i + 1 ) / ( 4 * cycleLength + 2 ) );
        omegas[i] = omega / ( 2 * cosval * cosval );
    }
    std::vector<int> lejaorder ( cycleLength );
    float maxVal = 0.f;
    for ( int i = 0; i < cycleLength; ++i )
        if ( 1/omegas[i] > maxVal ) {
            lejaorder[0] = i;
            maxVal = 1/omegas[i];
        }

    for ( int i = 1; i < cycleLength; ++i ) {
        maxVal = 0.f;
        for ( int j = 0; j < cycleLength; ++j ) {
            float prod = 1.f;
            for ( int k = 0; k < i; ++k )
                prod *= abs ( 1/omegas[j] - 1/omegas[lejaorder[k]] );
            if ( prod > maxVal ) {
                lejaorder[i] = j;
                maxVal = prod;
            }
        }
    }

    for ( int i = 0; i < 5000; ++i ) {
        buf1.setTo ( 0 );
        buf2.setTo ( 0 );
        for ( int j = 0; j < cycleLength; ++j ) {
            size_t shmem_sz =
                ( threads3.x + 4 ) * ( threads3.y + 4 ) * ( threads3.z + 4 )
                * sizeof ( float );
            kernel_multB<<<blocks3, threads3, shmem_sz>>> (
                sdf, w, dHull, dInter, buf1, volSize, voxelSize, alpha,
                betaHull, betaInter );
            hipDeviceSynchronize();
            cv::cuda::subtract ( c, buf1, buf1 );
            kernel_multDinv<<<blocks, threads>>> (
                sdf, buf1, w, dHull, dInter, volSize, voxelSize, alpha,
                betaHull, betaInter );
            hipDeviceSynchronize();
            cv::cuda::multiply ( omegas[lejaorder[j]], buf1, buf1 );
            cv::cuda::add ( sdf, buf1, sdf );

            cv::cuda::add ( buf2, buf1, buf2 );
        }
        float incrNorm = cv::cuda::norm ( buf2, cv::NORM_L2 )
                         / ( cycleLength * volSize.x * volSize.y * volSize.z );
        if ( incrNorm < 1e-10 ) {
            std::cout << std::endl << "Converged in iteration " << i
                      << " with an increment norm of " << incrNorm << std::endl;
            break;
        }

        std::cout << "\r" << std::setfill ( '0' ) << std::setw ( 4 ) << i
                  << ": " << std::scientific << std::setprecision ( 5 )
                  << incrNorm << std::flush;
    }

    std::cout.unsetf ( std::ios_base::floatfield );
    std::cout << std::endl;
}

__global__
void kernel_compIntersec ( const cv::cuda::PtrStep<float> otherC,
                           cv::cuda::PtrStepSz<float> d, const float33 rot,
                           const float3 trans, const float thisVoxelSize,
                           const float otherVoxelSize, const int3 thisRes,
                           const int3 otherRes ) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y_ = blockIdx.y * blockDim.y + threadIdx.y;

    if ( x >= d.cols || y_ >= d.rows )
        return;

    const int y = y_ % thisRes.y;
    const int z = y_ / thisRes.y;

    const float3 v = make_float3 (
                         ( x - ( thisRes.x - 1 ) / 2.f ) * thisVoxelSize,
                         ( y - ( thisRes.y - 1 ) / 2.f ) * thisVoxelSize,
                         ( z - ( thisRes.z - 1 ) / 2.f ) * thisVoxelSize );
    const float3 v_other = rot * v + trans;
    const float3 idx_other = ( v_other / otherVoxelSize )
                             + ( otherRes - 1 ) / 2.f;

    if ( idx_other.x < 0 || idx_other.x > otherRes.x - 2
            || idx_other.y < 0 || idx_other.y > otherRes.y - 2
            || idx_other.z < 0 || idx_other.z > otherRes.z - 2 )
        return;

    const float sdf = emf::cuda::TSDF::interpolateTrilinear ( otherC, idx_other,
                                                              otherRes );

    if ( sdf < 0.f ) {
        d ( y_, x ) = fmaxf ( d ( y_, x ), -sdf );
    }
}

void compIntersec ( const cv::cuda::GpuMat& otherC, cv::cuda::GpuMat& d,
                    const cv::Matx33f& rel_rot, const cv::Vec3f& rel_trans,
                    const float thisVoxelSize, const float otherVoxelSize,
                    const cv::Vec3i& thisVolumeRes,
                    const cv::Vec3i& otherVolumeRes ) {
    dim3 threads ( 32, 32 );
    dim3 blocks ( ( d.cols + threads.x - 1 ) / threads.x,
                  ( d.rows + threads.y - 1 ) / threads.y );

    const float33 rot = * ( float33 * ) rel_rot.val;
    const float3 trans = * ( float3 * ) rel_trans.val;

    const int3 thisRes = * ( int3 * ) thisVolumeRes.val;
    const int3 otherRes = * ( int3 * ) otherVolumeRes.val;

    kernel_compIntersec<<<blocks, threads>>> (
        otherC, d, rot, trans, thisVoxelSize, otherVoxelSize, thisRes,
        otherRes );
    hipDeviceSynchronize();
}

__global__
void kernel_compHull ( const cv::cuda::PtrStepSz<float> depth,
                       cv::cuda::PtrStep<float> d, cv::cuda::PtrStep<float> w,
                       cv::cuda::PtrStep<float> c,
                       cv::cuda::PtrStep<float> wcount, const float33 rot_OC,
                       const float3 trans_OC, const float33 intr,
                       const float sigma, const int3 volSize,
                       const float voxelSize ) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y_ = blockIdx.y * blockDim.y + threadIdx.y;

    if ( x >= volSize.x || y_ >= volSize.y * volSize.z )
        return;

    const int y = y_ % volSize.y;
    const int z = y_ / volSize.y;

    const float3 pos_obj = make_float3 (
                               ( x - ( volSize.x - 1 ) / 2.f ) * voxelSize,
                               ( y - ( volSize.y - 1 ) / 2.f ) * voxelSize,
                               ( z - ( volSize.z - 1 ) / 2.f ) * voxelSize );
    const float3 pos_cam = rot_OC * pos_obj + trans_OC;

    if ( pos_cam.z <= 0.f ) {
        return;
    }

    const float3 proj = intr * pos_cam;

    const int2 pix = make_int2 ( __float2int_rn ( proj.x / proj.z ),
                                 __float2int_rn ( proj.y / proj.z ) );

    if ( pix.x < 0 || pix.x >= depth.cols || pix.y < 0 || pix.y >= depth.rows )
        return;

    const float depthVal = depth ( pix.y, pix.x );
    if ( depthVal <= 0.f ) {
        return;
    }

    const float lambda = norm ( make_float3 (
                                    ( pix.x - intr ( 0, 2 ) ) / intr ( 0, 0 ),
                                    ( pix.y - intr ( 1, 2 ) ) / intr ( 1, 1 ),
                                    1.f ) );

    const float sdf = depthVal - ( 1.f / lambda ) * norm ( pos_cam );

    if ( sdf > 3*sigma*voxelSize ) {
        d ( y_, x ) = fmaxf ( d ( y_, x ), voxelSize );
        w ( y_, x ) = 0.f;
        c ( y_, x ) = 0.f;
        wcount ( y_, x ) = 0.f;
    }
}

void compHull ( const cv::cuda::GpuMat& depthMap, cv::cuda::GpuMat& d,
                cv::cuda::GpuMat& w, cv::cuda::GpuMat& c,
                cv::cuda::GpuMat& wcount, const cv::Matx33f& rel_rot_OC,
                const cv::Vec3f& rel_trans_OC, const cv::Matx33f& intr,
                const float sigma, const cv::Vec3i& volumeRes,
                const float voxelSize ) {
    dim3 threads ( 32, 32 );
    dim3 blocks ( ( d.cols + threads.x - 1 ) / threads.x,
                  ( d.rows + threads.y - 1 ) / threads.y );

    const int3 volSize = * ( int3 * ) volumeRes.val;

    const float33 rot = * ( float33 * ) rel_rot_OC.val;
    const float3 trans = * ( float3 * ) rel_trans_OC.val;

    const float33 camIntr = * ( float33 * ) intr.val;

    kernel_compHull<<<blocks, threads>>> (
        depthMap, d, w, c, wcount, rot, trans, camIntr, sigma, volSize,
        voxelSize );
    hipDeviceSynchronize();
}

__global__
void kernel_upSample ( const cv::cuda::PtrStepSz<float> lowRes,
                       cv::cuda::PtrStep<float> highRes, const int3 lowVolRes,
                       const int3 highVolRes ) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y_ = blockIdx.y * blockDim.y + threadIdx.y;

    if ( x >= lowRes.cols || y_ >= lowRes.rows )
        return;

    const int y = y_ % lowVolRes.y;
    const int z = y_ / lowVolRes.y;

    float val = lowRes ( y_, x );

    highRes ( z * 2 * highVolRes.y + y * 2, x * 2 ) = val;
    highRes ( z * 2 * highVolRes.y + y * 2, x * 2 + 1 ) = val;
    highRes ( z * 2 * highVolRes.y + y * 2 + 1, x * 2 ) = val;
    highRes ( z * 2 * highVolRes.y + y * 2 + 1, x * 2 + 1 ) = val;
    highRes ( ( z * 2 + 1 ) * highVolRes.y + y * 2, x * 2 ) = val;
    highRes ( ( z * 2 + 1 ) * highVolRes.y + y * 2, x * 2 + 1 ) = val;
    highRes ( ( z * 2 + 1 ) * highVolRes.y + y * 2 + 1, x * 2 ) = val;
    highRes ( ( z * 2 + 1 ) * highVolRes.y + y * 2 + 1, x * 2 + 1 ) = val;
}

void upSample ( const cv::cuda::GpuMat& lowRes, cv::cuda::GpuMat& highRes,
                const cv::Vec3i& lowVolumeRes, const cv::Vec3i& highVolumeRes
              ) {
    dim3 threads ( 32, 32 );
    dim3 blocks ( ( lowRes.cols + threads.x - 1 ) / threads.x,
                  ( lowRes.rows + threads.y - 1 ) / threads.y );

    int3 lowVolRes = * ( int3 * ) lowVolumeRes.val;
    int3 highVolRes = * ( int3 * ) highVolumeRes.val;

    kernel_upSample<<<blocks, threads>>> (
        lowRes, highRes, lowVolRes, highVolRes );
    hipDeviceSynchronize();
}

}
}
}
