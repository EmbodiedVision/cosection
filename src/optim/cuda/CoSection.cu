#include "hip/hip_runtime.h"
/*
 * This file is part of Co-Section.
 *
 * Copyright (C) 2020 Embodied Vision Group, Max Planck Institute for Intelligent Systems, Germany.
 * Developed by Michael Strecke <mstrecke at tue dot mpg dot de>.
 * For more information see <https://cosection.is.tue.mpg.de/>.
 * If you use this code, please cite the respective publication as
 * listed on the website.
 *
 * Co-Section is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * Co-Section is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with Co-Section.  If not, see <https://www.gnu.org/licenses/>.
 */
#include "CoSection/optim/cuda/CoSection.cuh"

using emf::cuda::operator-;
using emf::cuda::operator/;
using emf::cuda::operator==;
using emf::cuda::norm;
using emf::cuda::cross;

namespace cosection {
namespace cuda {
namespace CoSection {

__global__
void kernel_computeNormals ( const cv::cuda::PtrStepSz<float3> points,
                             cv::cuda::PtrStep<float3> normals ) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ( x < 1 || x >= points.cols - 1 || y < 1 || y >= points.rows - 1
            || points ( y, x + 1 ).z == 0.f || points ( y, x - 1 ).z == 0
            || points ( y + 1, x ).z == 0.f || points ( y - 1, x ).z == 0
            || points ( y, x ) == 0.f )
        return;

    const float3 dx = ( points ( y, x + 1 ) - points ( y, x - 1 ) );
    const float3 dy = ( points ( y + 1, x ) - points ( y - 1, x ) );
    const float3 dir = cross ( dx, dy );

    const float3 normal = dir / norm ( dir );

    normals ( y, x ) = normal.z > 0 ? -normal : normal;
}

void computeNormals ( const cv::cuda::GpuMat& points,
                      cv::cuda::GpuMat& normals ) {
    // TODO: find good thread/block parameters
    dim3 threads ( 32, 32 );
    dim3 blocks ( ( points.cols + threads.x - 1 ) / threads.x,
                  ( points.rows + threads.y - 1 ) / threads.y );
    normals.setTo ( cv::Scalar::all ( 0.f ) );

    kernel_computeNormals<<<blocks, threads>>> ( points, normals );
    hipDeviceSynchronize();
}

}
}
}
